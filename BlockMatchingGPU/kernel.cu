#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>

#include <cstdio>

#define FRAMES 30
#define HEIGHT 320
#define WIDTH 480



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
int ***allocateVideo( int x, int y, int t );
int ***initializeVideo(int ***video, int x, int y, int t);



__global__ void parrallelBlockMatching()
{

}

__global__ int ***motionEstimation(int ***video, int ***motion, int x, int y, int t, int b, int r)
{
	int f;
	for(f=0;f<t-1;f++)
	{
		motion=motionFrame(video,motion,x,y,f,b,r);
	}
	return motion;
}
__global__ int ***motionFrame(int ***video, int ***motion, int x, int y, int t, int b, int r)
{
	int i,j;
	
	for(i=r;i<=x-b-r;i++)
	{
		for(j=r;j<=y-b-r;j++)
		{
			motion[t][i][j]=blockMatching(video,i,j,t,b,r);
			//to optimize: make i and j increment by b instead of 1
		}
	}
	
	return motion;
}
__global__ int blockMatching(int ***video, int x, int y, int t, int b, int r)
{
	int diff, best_diff, i, j;
	double best_dist,dist;
	best_diff=1000000;
	
	for(i=x-r;i<=x+r;i++)
	{
		for(j=y-r;j<=y+r;j++)
		{
			diff=blockDiff(video,x,y,t,i,j,b);
			
			if(diff<best_diff)
			{
				best_diff=diff;
				best_dist=((x-i)*(x-i)+(y-j)*(y-j));
			}
			else if(diff==best_diff)
			{
				dist=((x-i)*(x-i)+(y-j)*(y-j));
				if(dist<best_dist)
				{
					best_dist=dist;
				}
			}
		}
	}
	
	return best_dist;
}
__global__ int blockDiff(int ***video, int x, int y, int t, int cx, int cy, int b)
{
	int i,j;
	int diff=0;
	
	for(i=0;i<b;i++)
	{
		for(j=0;j<b;j++)
		{
			diff+=abs(video[t][x+i][y+j]-video[t+1][cx+i][cy+j]);
			//to optimize: break when diff exceed best_diff
		}
	}
	
	return diff;
}

int main()
{
	// Allocate and Initialize video 
	int ***video = allocateVideo(WIDTH,HEIGHT,FRAMES);
	int ***motionVideo = allocateVideo(WIDTH,HEIGHT,FRAMES);

	video = initializeVideo(video,WIDTH,HEIGHT,FRAMES);

	// Allocate and initialize in device
	int *** d_video;
	int *** d_motionVideo;
	d_video = (int***) hipMalloc(&d_video,sizeof(int)*FRAMES*WIDTH*HEIGHT);
	d_motionVideo = (int***) hipMalloc(&d_motionVideo,sizeof(int)*FRAMES*WIDTH*HEIGHT);
	
	hipMemcpy(d_video,&video,sizeof(int)*FRAMES*WIDTH*HEIGHT, hipMemcpyHostToDevice);
	hipMemcpy(d_motionVideo,&motionVideo,sizeof(int)*FRAMES*WIDTH*HEIGHT, hipMemcpyHostToDevice);
	

	// perform serial computations in device
	motionEstimation <<< 1,1 >>> (d_video,d_motionVideo,WIDTH,HEIGHTFRAMES);
	// perform parallel computations in device 
	hipFree(d_video);
	free(video);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.

int ***allocateVideo( int x, int y, int t )
{
  int i,j;
  int ***video;
  video = (int ***) malloc(t * sizeof(int**));
  for (i=0;i<t;i++)
  {
	video[i] = (int **) malloc( x * sizeof(int*));
	for (j=0;j<x;j++)
	  video[i][j] = (int *) malloc( y * sizeof(int));
  }
  return video;
}

int ***initializeVideo(int ***video, int x, int y, int t)
{
  int i,j,l,l_f;
  
  l=y/2;
  l_f=1;
  for (i=0;i<t;i++)
  {
    l_f=rand()%5-2;
	l+=l_f;
	if(l<2) l=1;
	if(l>y-4) l=y-4;
	
	for (j=1;j<x-1;j++)
	{
		video[i][j][l] = (j+15)%256;
		video[i][j][l+1] = (j+16)%256;
		video[i][j][l+2] = (j+15)%256;
	}
  }
  return video;
}


